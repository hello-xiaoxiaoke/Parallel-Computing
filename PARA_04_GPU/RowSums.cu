#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// CUDA example:  finds row sums of an integer matrix m

// find1elt() finds the row sum of one row of the nxn matrix m, 
// storing the result in the corresponding position in the 
// rowsum array rs; matrix is in 1-dimensional, row-major order

// this is the "kernel", which each thread on the GPU executes
__global__ void find1elt(int *m, int *rs, int n)
{
   // this thread will handle row # rownum
   int rownum = blockIdx.x;  
   int sum = 0;
   for (int k = 0; k < n; k++)
      sum += m[rownum*n+k];
   rs[rownum] = sum;
}

// the remaining code is executed on the CPU
int main(int argc, char **argv)
{
    int n = atoi(argv[1]);  // number of matrix rows/cols
    int *hm, // host matrix
        *dm, // device matrix
        *hrs, // host rowsums
        *drs; // device rowsums
    // size of matrix in bytes
    int msize = n * n * sizeof(int);  
    // allocate space for host matrix
    hm = (int *) malloc(msize);  
    // as a test, fill matrix with consecutive integers
    int t = 0,i,j;
    for (i = 0; i < n; i++) {
       for (j = 0; j < n; j++) {
          hm[i*n+j] = t++;
       }
    }
    // allocate matrix space at device 
    hipMalloc((void **)&dm,msize);
    // copy host matrix to device matrix
    hipMemcpy(dm,hm,msize,hipMemcpyHostToDevice);
    // allocate host, device rowsum arrays
    int rssize = n * sizeof(int);
    hrs = (int *) malloc(rssize);  
    hipMalloc((void **)&drs,rssize);
    // set up threads structure parameters
    dim3 dimGrid(n,1);  // n blocks in the grid
    dim3 dimBlock(1,1,1);  // 1 thread per block
    // launch the kernel
    find1elt<<<dimGrid,dimBlock>>>(dm,drs,n);
    // wait until kernel finishes
    hipDeviceSynchronize();
    // copy row vector from device to host
    hipMemcpy(hrs,drs,rssize,hipMemcpyDeviceToHost);
    // check results
    if (n < 10) for(int i=0; i<n; i++) printf("%d\n",hrs[i]);
    // clean up, very important 
    free(hm);
    hipFree(dm);
    free(hrs);
    hipFree(drs);
}
